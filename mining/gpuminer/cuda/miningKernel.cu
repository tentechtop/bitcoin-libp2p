#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdbool.h>
#include <string.h>
#include <stdio.h>

// 结果数据结构（供主机端读取）
typedef struct {
    int found;          // 1=找到有效哈希，0=未找到
    uint32_t nonce;     // 有效nonce或最后尝试的nonce
    uint8_t hash[32];   // 对应的哈希值
} MiningResult;

// 全局变量：存储计算结果（设备端）
__device__ MiningResult devResult;

// 工具函数：字节序反转
__device__ void reverseBytes(uint8_t *data, int length) {
    for (int i = 0; i < length / 2; i++) {
        uint8_t temp = data[i];
        data[i] = data[length - 1 - i];
        data[length - 1 - i] = temp;
    }
}

// SHA-256常量
__constant__ uint32_t k[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

// SHA-256辅助函数
__device__ uint32_t rotr(uint32_t x, int n) {
    return (x >> n) | (x << (32 - n));
}

__device__ uint32_t ch(uint32_t x, uint32_t y, uint32_t z) {
    return (x & y) ^ (~x & z);
}

__device__ uint32_t maj(uint32_t x, uint32_t y, uint32_t z) {
    return (x & y) ^ (x & z) ^ (y & z);
}

__device__ uint32_t sigma0(uint32_t x) {
    return rotr(x, 2) ^ rotr(x, 13) ^ rotr(x, 22);
}

__device__ uint32_t sigma1(uint32_t x) {
    return rotr(x, 6) ^ rotr(x, 11) ^ rotr(x, 25);
}

__device__ uint32_t gamma0(uint32_t x) {
    return rotr(x, 7) ^ rotr(x, 18) ^ (x >> 3);
}

__device__ uint32_t gamma1(uint32_t x) {
    return rotr(x, 17) ^ rotr(x, 19) ^ (x >> 10);
}

// SHA-256哈希计算（输入数据，长度，输出哈希）
__device__ void sha256(uint8_t *data, int len, uint8_t *hash) {
    uint32_t h[8] = {
        0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
        0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19
    };

    int numBlocks = (len + 8 + 63) / 64;  // 计算需要的512位块数
    uint8_t block[64];                    // 单个块缓冲区

    for (int b = 0; b < numBlocks; b++) {
        memset(block, 0, 64);  // 初始化块

        // 复制数据到块
        int copyLen = len - b * 64;
        if (copyLen > 0) {
            if (copyLen > 64) copyLen = 64;
            memcpy(block, data + b * 64, copyLen);
        }

        // 填充（最后一个块）
        if (b == numBlocks - 1) {
            // 添加0x80标记
            if (copyLen < 64) {
                block[copyLen] = 0x80;
            }

            // 填充长度（bits）
            if (64 - copyLen >= 9) {  // 确保有空间存放8字节长度
                uint64_t bitsLen = (uint64_t)len * 8;
                for (int i = 0; i < 8; i++) {
                    block[64 - 8 + i] = (bitsLen >> (8 * (7 - i))) & 0xff;
                }
            }
        }

        // 消息调度（扩展为64个字）
        uint32_t w[64];
        for (int t = 0; t < 16; t++) {
            w[t] = (block[t*4] << 24) | (block[t*4+1] << 16) |
                   (block[t*4+2] << 8) | block[t*4+3];
        }
        for (int t = 16; t < 64; t++) {
            w[t] = gamma1(w[t-2]) + w[t-7] + gamma0(w[t-15]) + w[t-16];
        }

        // 压缩循环
        uint32_t a = h[0], b = h[1], c = h[2], d = h[3];
        uint32_t e = h[4], f = h[5], g = h[6], h_val = h[7];

        for (int t = 0; t < 64; t++) {
            uint32_t temp1 = h_val + sigma1(e) + ch(e, f, g) + k[t] + w[t];
            uint32_t temp2 = sigma0(a) + maj(a, b, c);
            h_val = g;
            g = f;
            f = e;
            e = d + temp1;
            d = c;
            c = b;
            b = a;
            a = temp1 + temp2;
        }

        // 更新哈希值
        h[0] += a; h[1] += b; h[2] += c; h[3] += d;
        h[4] += e; h[5] += f; h[6] += g; h[7] += h_val;
    }

    // 转换为字节数组（大端序）
    for (int i = 0; i < 8; i++) {
        hash[i*4]   = (h[i] >> 24) & 0xff;
        hash[i*4+1] = (h[i] >> 16) & 0xff;
        hash[i*4+2] = (h[i] >> 8) & 0xff;
        hash[i*4+3] = h[i] & 0xff;
    }
}

// 将4字节压缩难度目标转换为256位目标值
__device__ void compactToTarget(uint8_t *compact, uint8_t *target) {
    memset(target, 0, 32);  // 初始化目标值为0

    int exponent = compact[0] & 0xFF;  // 指数（无符号）
    uint8_t coefficient[3] = {compact[1], compact[2], compact[3]};  // 系数

    // 计算目标值在256位数组中的起始位置（右对齐）
    int startIdx = 32 - exponent;
    if (startIdx < 0) startIdx = 0;

    // 复制系数到目标值
    for (int i = 0; i < 3 && (startIdx + i) < 32; i++) {
        target[startIdx + i] = coefficient[i];
    }
}

// 检查哈希是否小于等于目标值
__device__ bool isHashValid(uint8_t *hash, uint8_t *target) {
    for (int i = 0; i < 32; i++) {
        if (hash[i] < target[i]) return true;   // 哈希更小，有效
        if (hash[i] > target[i]) return false;  // 哈希更大，无效
    }
    return true;  // 相等，有效
}

// 内核函数：处理序列化的区块头和nonce范围
extern "C" __global__ void findValidNonceGPU(
    uint8_t *serializedHeader,  // 序列化的区块头（80字节）
    uint32_t startNonce,        // 起始nonce
    uint32_t endNonce           // 结束nonce
) {
    // 初始化结果（仅第一个线程执行）
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        devResult.found = 0;
        devResult.nonce = endNonce;  // 默认最后一个nonce
        memset(devResult.hash, 0, 32);
    }
    __syncthreads();  // 等待初始化完成

    // 计算全局线程ID
    uint32_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t totalThreads = gridDim.x * blockDim.x;

    // 若已找到有效结果，直接返回
    if (devResult.found) return;

    // 计算当前线程负责的nonce范围
    uint32_t nonceRange = endNonce - startNonce + 1;
    uint32_t threadsPerNonce = (nonceRange + totalThreads - 1) / totalThreads;  // 向上取整
    uint32_t threadStart = startNonce + globalId * threadsPerNonce;
    uint32_t threadEnd = threadStart + threadsPerNonce - 1;
    if (threadEnd > endNonce) threadEnd = endNonce;

    // 若线程负责的范围无效，返回
    if (threadStart > endNonce) return;

    // 准备计算资源
    uint8_t headerCopy[80];  // 复制区块头（避免多线程冲突）
    memcpy(headerCopy, serializedHeader, 80);

    // 提取难度目标（序列化区块头的72-75字节，小端存储，需反转）
    uint8_t compactTarget[4];
    memcpy(compactTarget, headerCopy + 72, 4);  // 难度目标在序列化头中的位置
    reverseBytes(compactTarget, 4);  // 转为大端用于解析

    // 预计算目标值
    uint8_t target[32];
    compactToTarget(compactTarget, target);

    // 临时变量
    uint8_t hash1[32], hash2[32];  // 双重哈希结果
    uint32_t currentNonce;

    // 遍历线程负责的nonce范围
    for (currentNonce = threadStart; currentNonce <= threadEnd; currentNonce++) {
        // 若已找到结果，退出循环
        if (devResult.found) break;

        // 更新区块头中的nonce（最后4字节，小端存储）
        headerCopy[76] = (currentNonce >> 0) & 0xFF;  // nonce第1字节（小端）
        headerCopy[77] = (currentNonce >> 8) & 0xFF;   // nonce第2字节
        headerCopy[78] = (currentNonce >> 16) & 0xFF;  // nonce第3字节
        headerCopy[79] = (currentNonce >> 24) & 0xFF;  // nonce第4字节

        // 双重SHA-256计算
        sha256(headerCopy, 80, hash1);  // 第一次哈希
        sha256(hash1, 32, hash2);       // 第二次哈希
        // 新增：反转哈希字节序，与CPU端保持一致
        reverseBytes(hash2, 32);  // 关键修改：反转32字节哈希

        // 检查哈希是否有效
        if (isHashValid(hash2, target)) {
            // 原子操作：确保只有第一个找到的结果被记录
            int expected = 0;
            if (atomicCAS(&devResult.found, expected, 1) == expected) {
                devResult.nonce = currentNonce;
                memcpy(devResult.hash, hash2, 32);
            }
            break;  // 找到后退出循环
        }
    }

    // 记录最后尝试的nonce和哈希（仅当未找到有效结果时）
    if (!devResult.found && currentNonce == threadEnd + 1) {
        // 原子操作：确保最后一个nonce被正确记录（取最大的nonce）
        atomicMax((uint32_t*)&devResult.nonce, currentNonce - 1);
        // 仅最后一个线程更新哈希（确保是最后一个nonce的哈希）
        if (globalId == totalThreads - 1) {
            sha256(headerCopy, 80, hash1);
            sha256(hash1, 32, hash2);
            memcpy(devResult.hash, hash2, 32);
        }
    }
}

// 主机端函数：初始化设备并执行内核（供CUDA C调用，Java通过JCUDA间接调用）
extern "C" void launchMining(
    uint8_t *hostHeader,    // 主机端序列化的区块头（80字节）
    uint32_t startNonce,    // 起始nonce
    uint32_t endNonce,      // 结束nonce
    MiningResult *hostResult,  // 主机端结果指针
    int blockSize,          // 线程块大小
    int gridSize            // 网格大小
) {
    // 设备端变量
    uint8_t *d_header;
    MiningResult *d_result;

    // 分配设备内存
    hipMalloc(&d_header, 80);  // 区块头固定80字节
    hipMalloc(&d_result, sizeof(MiningResult));

    // 复制区块头到设备
    hipMemcpy(d_header, hostHeader, 80, hipMemcpyHostToDevice);

    // 启动内核
    findValidNonceGPU<<<gridSize, blockSize>>>(d_header, startNonce, endNonce);
    hipDeviceSynchronize();  // 等待内核执行完成

    // 复制结果到主机
    hipMemcpyFromSymbol(hostResult, HIP_SYMBOL(devResult), sizeof(MiningResult));

    // 释放设备内存
    hipFree(d_header);
    hipFree(d_result);
}